//
// Created by gaborn on 2024-01-07.
//
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "kmeans_cuda_kernel.cuh"
#include "alloc.h"

extern "C"
void allocateFeatures(float **features, int npoints, int nfeatures) {
    gpuCheck(hipMallocManaged(features, npoints * nfeatures * sizeof(float)));
    gpuCheck(hipMemPrefetchAsync(features[0], npoints * nfeatures * sizeof(float), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchFeaturesToHost(float *features, int npoints, int nfeatures) {
    gpuCheck(hipMemPrefetchAsync(features, npoints * nfeatures * sizeof(float), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchFeaturesToDevice(float *features, int npoints, int nfeatures) {
    gpuCheck(hipMemPrefetchAsync(features, npoints * nfeatures * sizeof(float), 0));
}

extern "C"
void deallocateFeatures(float *features) {
    gpuCheck(hipFree(features));
}

extern "C"
void allocateClusters(float **clusters, int nclusters, int nfeatures) {
    gpuCheck(hipMallocManaged(clusters, nclusters * nfeatures * sizeof(float)));
    gpuCheck(hipMemPrefetchAsync(clusters[0], nclusters * nfeatures * sizeof(float), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchClustersToHost(float *clusters, int nclusters, int nfeatures) {
    gpuCheck(hipMemPrefetchAsync(clusters, nclusters * nfeatures * sizeof(float), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchClustersToDevice(float *clusters, int nclusters, int nfeatures) {
    gpuCheck(hipMemPrefetchAsync(clusters, nclusters * nfeatures * sizeof(float), 0));
}

extern "C"
void deallocateClusters(float *clusters) {
    gpuCheck(hipFree(clusters));
}

extern "C"
void allocateMembership(int **membership, int npoints) {
    gpuCheck(hipMallocManaged(membership, npoints * sizeof(int)));
    gpuCheck(hipMemPrefetchAsync(membership[0], npoints * sizeof(int), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchMembershipToHost(int *memberhsip, int npoints) {
    gpuCheck(hipMemPrefetchAsync(memberhsip, npoints * sizeof(int), hipCpuDeviceId));
    gpuCheck(hipDeviceSynchronize());
}

extern "C"
void prefetchMembershipToDevice(int *memberhsip, int npoints) {
    gpuCheck(hipMemPrefetchAsync(memberhsip, npoints * sizeof(int), 0));
}

extern "C"
void deallocateMembership(int *membership) {
    gpuCheck(hipFree(membership));
}
